#include "hip/hip_runtime.h"
#include <stdio.h>

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

__global__
void initWith(float num, float *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = idx; i < N; i+=stride)
  {
    a[idx] = num;
  }
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("\nFAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("\nSuccess! All values calculated correctly.\n");
}

int main()
{
  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
  
  int deviceId;
  hipGetDevice(&deviceId);

  // Prefetching the vectors
  hipMemPrefetchAsync(a, size, deviceId);
  // hipMemPrefetchAsync(b, size, deviceId);
  // hipMemPrefetchAsync(c, size, deviceId);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 800;

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);
  int SMs = props.multiProcessorCount;
  // The lines above can be changed by using "hipDeviceGetAttribute(pointer_to_store, attributeName, deviceId)"
  // hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  // printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);

  numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

  numberOfBlocks = numberOfBlocks + (SMs - numberOfBlocks % SMs);
  // numberOfBlocks = (((N + threadsPerBlock - 1) / threadsPerBlock) + SMs - 1) / SMs;
  printf("------------------------------");
  printf("\nVector length is: %d", N);
  printf("\nNumber of threads per block is: %d", threadsPerBlock);
  printf("\nNumber of blocks is: %d", numberOfBlocks);

  initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);


  /*
   * nsys should register performance changes when execution configuration
   * is updated.
   */


  hipError_t addVectorsErr;
  hipError_t asyncErr;

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("\nError: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("\nError: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
